#include "hip/hip_runtime.h"
#include "golCUDA.h"

__global__ void GolKernel_random(unsigned int* cellData, float density, int seed)
{
	unsigned int xId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int yId = threadIdx.y + blockIdx.y * blockDim.y;

	unsigned int idx = gridDim.x * blockDim.x * yId + xId;

	hiprandState localState;
	hiprand_init(seed, idx, 0, &localState);

	cellData[idx] = (hiprand_uniform(&localState) > density ? 1 : 0);
}

__global__ void GolKernel_next(unsigned int* cellData, unsigned int* cellNext)
{
	int xId = threadIdx.x + blockIdx.x * blockDim.x;
	int yId = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = gridDim.x * blockDim.x * yId + xId;

	unsigned int state = cellData[idx];
	int nextState;

	if ((xId < (gridDim.x * blockDim.x)) && (xId > 0) && (yId > 0) && (yId < (gridDim.y * blockDim.y)))
	{
		unsigned int sum = 0;
		for (int i = 0; i < 3; i++)
		{
			for (int j = 0; j < 3; j++)
			{
				idx = gridDim.x * blockDim.x * (yId + j - 1) + (xId + i - 1);
				sum += cellData[idx];
				//std::printf("%d", sum);
			}
		}
		sum -= state;
		nextState = ( sum==3 || (state==1 && sum==2) );
	}
	else {
		nextState = 0;
	}

	idx = gridDim.x * blockDim.x * yId + xId;
	cellNext[idx] = nextState;

}

__global__ void GolKernel_copy(unsigned int* cellData, unsigned int* cellNext)
{
	int xId = threadIdx.x + blockIdx.x * blockDim.x;
	int yId = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = gridDim.x * blockDim.x * yId + xId;

	cellData[idx] = cellNext[idx];
}
