#include "hip/hip_runtime.h"
//
// Created by conor on 26/10/24.
//

#include "MaxwellPipeline.h"
#include <iostream>

MaxwellPipeline::MaxwellPipeline(dim3 threads, dim3 blocks, unsigned int widthX, unsigned int widthY)
    : m_threads(threads), m_blocks(blocks), m_widthX(widthX), m_widthY(widthY) {
}

MaxwellPipeline::~MaxwellPipeline() {
    glDisableVertexAttribArray(0);
}

bool MaxwellPipeline::Init() {
    GLint ret = true;
    GLclampf Red = 0.0f, Green = 0.0f, Blue = 0.0f, Alpha = 0.0f;

    glClearColor(Red, Green, Blue, Alpha);

    // charges
    float chargePos[] = {
        -0.5f, 0.0f,
        0.5f, 0.0f
    };

    // For the number of pixel, initialise a field which is simply 0.0 everywhere
    float fieldStrength[m_widthX * m_widthY];
    for (int i = 0; i < m_widthX * m_widthY; i++) {
        fieldStrength[i] = 0.0f;
    }

    m_BufferSizeField = m_widthX * m_widthY * sizeof(float);
    m_BufferSizeCharges = 2 * 2 * sizeof(float);

    // Init Field Buffer
    glGenBuffers(1, &m_VBOField);
    glBindBuffer(GL_ARRAY_BUFFER, m_VBOField);
    glBufferData(GL_ARRAY_BUFFER, m_widthX * m_widthY * sizeof(float), fieldStrength, GL_DYNAMIC_DRAW);

    // Init Charges Buffer
    glGenBuffers(1, &m_VBOCharges);
    glBindBuffer(GL_ARRAY_BUFFER, m_VBOCharges);
    glBufferData(GL_ARRAY_BUFFER, 2 * 2 * sizeof(float), chargePos, GL_DYNAMIC_DRAW);

    // Shaders
    m_shaderField = Shader("/home/conor/dev/CUDA-GameOfLife/shaders/Field.shader");
    m_shaderCharges = Shader("/home/conor/dev/CUDA-GameOfLife/shaders/Charges.shader");

    // CUDA graphics resource
    hipGraphicsGLRegisterBuffer(&m_resourceField, m_VBOField, hipGraphicsRegisterFlagsNone);
    hipGraphicsGLRegisterBuffer(&m_resourceCharges, m_VBOCharges, hipGraphicsRegisterFlagsNone);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    return ret;
}

void MaxwellPipeline::Draw() {
    m_shaderField.Bind();
    // set uniforms
    m_shaderField.SetUniformUint("xWidth", m_widthX);
    m_shaderField.SetUniformUint("yWidth", m_widthY);
    // Field
    glBindBuffer(GL_ARRAY_BUFFER, m_VBOField);
    // Attrib
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 1, GL_FLOAT, GL_FALSE, sizeof(float), (void *) 0);

    glDrawArrays(GL_POINTS, 0, m_widthX * m_widthY);

    glDisableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    m_shaderField.Unbind();
    m_shaderCharges.Bind();
    // bind buffers
    glBindBuffer(GL_ARRAY_BUFFER, m_VBOCharges);
    // Attrib
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 2 * sizeof(float), (void *) 0);

    // Draw
    glDrawArrays(GL_POINTS, 0, 2);

    // unbnd
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glDisableVertexAttribArray(0);
    m_shaderCharges.Unbind();
}

// Update field kernel
__global__ void UpdateFieldKernel(float *field, float *charges, unsigned int chargeCount, unsigned int widthX,
                                  unsigned int widthY) {
    int xId = threadIdx.x + blockIdx.x * blockDim.x;
    int yId = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = yId * widthX + xId;

    if (xId >= widthX || yId >= widthY) return;

    // positions on a scale of -1 to 1
    float x = (2.0f * xId / (widthX - 1)) - 1.0f;
    float y = (2.0f * yId / (widthY - 1)) - 1.0f;

    float fieldMagnitude = 0.0f;
    field[idx] = 0.;
    // d
    for (unsigned int i = 0; i < chargeCount; i++) {
        float chargeX = charges[2 * i];
        float chargeY = charges[2 * i + 1];

        float dx = chargeX - x;
        float dy = chargeY - y;
        float distanceSquared = dx * dx + dy * dy;
        float distance = sqrt(distanceSquared);
        float fieldStrength = 1.0f / distanceSquared; // Coulomb's law (assuming unit charge)

        field[idx] += fieldStrength;
    }
}

void MaxwellPipeline::UpdateField() {
    float *m_DevField;
    hipGraphicsMapResources(1, &m_resourceField, 0);
    hipGraphicsResourceGetMappedPointer((void **) &m_DevField, &m_BufferSizeField, m_resourceField);

    float *m_DevCharges;
    hipGraphicsMapResources(1, &m_resourceCharges, 0);
    hipGraphicsResourceGetMappedPointer((void **) &m_DevCharges, &m_BufferSizeCharges, m_resourceCharges);

    // Call CUDA kernel to update field
    UpdateFieldKernel<<<m_blocks, m_threads>>>(m_DevField, m_DevCharges, 2, m_widthX, m_widthY);

    hipGraphicsUnmapResources(1, &m_resourceField, 0);
    hipGraphicsUnmapResources(1, &m_resourceCharges, 0);
}


void MaxwellPipeline::UpdateCharges() {
    float *m_DevCharges;
    hipGraphicsMapResources(1, &m_resourceCharges, 0);
    hipGraphicsResourceGetMappedPointer((void **) &m_DevCharges, &m_BufferSizeCharges, m_resourceCharges);

    // Call CUDA kernel to update charges
    // Example: UpdateChargesKernel<<<m_blocks, m_threads>>>(m_DevCharges);

    hipGraphicsUnmapResources(1, &m_resourceCharges, 0);
}

void MaxwellPipeline::Reset() {
    float *m_DevField;
    float *m_DevCharges;
    hipGraphicsMapResources(1, &m_resourceField, 0);
    hipGraphicsResourceGetMappedPointer((void **) &m_DevField, &m_BufferSizeField, m_resourceField);
    hipGraphicsMapResources(1, &m_resourceCharges, 0);
    hipGraphicsResourceGetMappedPointer((void **) &m_DevCharges, &m_BufferSizeCharges, m_resourceCharges);

    // Call CUDA kernel to reset field and charges
    // Example: ResetFieldKernel<<<m_blocks, m_threads>>>(m_DevField);
    // Example: ResetChargesKernel<<<m_blocks, m_threads>>>(m_DevCharges);

    hipGraphicsUnmapResources(1, &m_resourceField, 0);
    hipGraphicsUnmapResources(1, &m_resourceCharges, 0);
}
