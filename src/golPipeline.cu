#include "Utility.h"

#include "golPipeline.h"
#include <iostream>

bool GoLPipeline::Init()
{
    GLint ret = true;  
    GLclampf Red = 0.0f, Green = 0.0f, Blue = 1.0f, Alpha = 0.0f;

    glClearColor(Red, Green, Blue, Alpha);
    glPointSize(m_pointSize);
    
    // // Init Buffer
    glGenBuffers(1, &m_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, m_VBO);
    glBufferData(GL_ARRAY_BUFFER, m_widthX * m_widthY * sizeof(unsigned int), 0, GL_DYNAMIC_DRAW);

    // Attrib Pointer
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 1, GL_UNSIGNED_INT, GL_FALSE, 0, 0);

    // ShaderS
    m_shader = Shader("shaders/GameOfLife.shader");
    m_shader.Bind();
	m_shader.SetUniformUint("widthX", m_widthX);
    m_shader.SetUniformUint("widthY", m_widthY);
	m_shader.SetUniform4f("u_OnColour", 1., 1., 1., 1.);
	m_shader.SetUniform4f("u_OffColour", 0., 0., 0., 1.);
	m_shader.SetUniform4f("windowXY", -1.0, 1.0, -1.0, 1.0);

    // CUDA graphics resource 
    hipGraphicsGLRegisterBuffer(&m_resource, m_VBO, hipGraphicsRegisterFlagsNone);

    unsigned int* m_DevState;
    m_BufferSize = m_widthX * m_widthY * sizeof(unsigned int);
	hipGraphicsMapResources(1, &m_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&m_DevState, &m_BufferSize, m_resource);
	GolKernel_random<<<m_blocks, m_threads>>>(m_DevState, 0.75f, 0);
	hipGraphicsUnmapResources(1, &m_resource, 0);

    return ret;
}

void GoLPipeline::Draw()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT );
    glDrawArrays(GL_POINTS, 0, m_widthX * m_widthY);
}

void GoLPipeline::Update()
{
    unsigned int* m_DevState;
	unsigned int* m_DevNextState;
	hipMalloc((void**)&m_DevNextState, m_BufferSize);

	hipGraphicsMapResources(1, &m_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&m_DevState, &m_BufferSize, m_resource);

	GolKernel_next<<<m_blocks, m_threads>>>(m_DevState, m_DevNextState);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipMemcpy(m_DevState, m_DevNextState, m_BufferSize, hipMemcpyDeviceToDevice) );

	hipGraphicsUnmapResources(1, &m_resource, 0);
	hipFree(m_DevNextState);
}